
#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    // Basic pointer usage
    int x = 42;
    int* ptr = &x;  // ptr holds address of x
    
    std::cout << "Value of x: " << x << std::endl;
    std::cout << "Address of x: " << &x << std::endl;
    std::cout << "Value of ptr: " << ptr << std::endl;
    std::cout << "Value pointed to by ptr: " << *ptr << std::endl;
    
    return 0;
}

// int value = 42;
  //  int* ptr1 = &value;
  //  int** ptr2 = &ptr1;
  //  int*** ptr3 = &ptr2; --> Multi level pointers
