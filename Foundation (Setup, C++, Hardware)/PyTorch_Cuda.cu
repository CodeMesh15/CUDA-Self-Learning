#include "hip/hip_runtime.h"
import torch

# Check CUDA availability
print(f"CUDA available: {torch.cuda.is_available()}")
print(f"CUDA version: {torch.version.cuda}")
print(f"Device count: {torch.cuda.device_count()}")

# Create tensors on GPU
x = torch.randn(1000, 1000, device='cuda')
y = torch.randn(1000, 1000, device='cuda')

# GPU matrix multiplication
result = torch.mm(x, y)
print(f"Result shape: {result.shape}")

# Compilation and Optimization

@torch.compile
def optimized_function(x, y):
    return torch.mm(x, y) + torch.sin(x)

# 30% speedup out of the box
x = torch.randn(1000, 1000, device='cuda')
y = torch.randn(1000, 1000, device='cuda')
result = optimized_function(x, y)
