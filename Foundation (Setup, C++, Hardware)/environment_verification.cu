// hello_cuda.cu
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void hello_kernel() {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    printf("Hello from thread %d\n", idx);
}

int main() {
    // Launch kernel with 1 block, 10 threads
    hello_kernel<<<1, 10>>>();
    
    // Wait for GPU to finish
    hipDeviceSynchronize();
    
    std::cout << "CUDA setup successful!" << std::endl;
    return 0;
}
